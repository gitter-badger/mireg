#include "hip/hip_runtime.h"
#include <mireg.h>
#include <pcl_visualizer.h>
#include <pcl_helper.h>
#include <transformation.h>
#include <registration.h>

//const float min_cell_size = 0.1;	// in metres
//const int hist_size = 100;	// sampling size for histogram

const float min_cell_size = 0.1;	// in metres
const int hist_size = 100;	// sampling size for histogram

bool file_read(std::string filename, std::vector<point>& cloud)
{
	std::ifstream file(filename);
	cloud.clear();
	if(file.is_open())
	{
		float frame; file>>frame;
		point temp; temp.get_point(file);
		while(!file.eof())
		{
			cloud.push_back(temp);
			temp.get_point(file);
		}
		file.close();
		return true;
	}
	else
	{
		return false;
	}
}

bool file_write(std::string filename, std::vector<long double>& euler_rep)
{
	std::ofstream file(filename);
	if(file.is_open())
	{
		for(int i=0; i<int(euler_rep.size()); ++i)
		{
			file<<euler_rep[i]<<" ";
		}
		file<<std::endl;
		return true;
	}
	else
	{
		return false;
	}
}

void print_transform(std::vector<std::vector<long double>>& transformation_mat)
{
	for(int i=0; i<4; ++i)
	{
		for(int j=0; j<4; ++j)
		{
			std::cout<<transformation_mat[i][j]<<"\t\t";
		}
		std::cout<<std::endl;
	}
}

void convert(std::vector<long double>& eu, std::vector<std::vector<long double>>& transform)
{
	long double x=eu[0], y=eu[1], z=eu[2], thetax=eu[3], thetay=eu[4], thetaz=eu[5];
	thetax*=(pi/180.0); thetay*=(pi/180.0);	thetaz*=(pi/180.0);

	transform[0][0]=cos(thetay)*cos(thetaz);
	transform[0][1]=sin(thetax)*sin(thetay)*cos(thetaz)-cos(thetax)*sin(thetaz);
	transform[0][2]=cos(thetax)*sin(thetay)*cos(thetaz)+sin(thetax)*sin(thetaz);
	
	transform[1][0]=cos(thetay)*sin(thetaz);
	transform[1][1]=sin(thetax)*sin(thetay)*sin(thetaz)+cos(thetax)*cos(thetaz);
	transform[1][2]=cos(thetax)*sin(thetay)*sin(thetaz)-sin(thetax)*cos(thetaz);
	
	transform[2][0]=-sin(thetay);
	transform[2][1]=sin(thetax)*cos(thetay);
	transform[2][2]=cos(thetax)*cos(thetay);
	
	transform[0][3]=x;
	transform[1][3]=y;
	transform[2][3]=z;
	transform[3][3]=1.0;
}

void process(std::vector<point>& scan, std::vector<std::vector<long double>>& transform)
{
	for(int i=0; i<int(scan.size()); ++i)
	{
		point temp=scan[i];
		temp.x=transform[0][0]*scan[i].x+transform[0][1]*scan[i].y+transform[0][2]*scan[i].z+transform[0][3];
		temp.y=transform[1][0]*scan[i].x+transform[1][1]*scan[i].y+transform[1][2]*scan[i].z+transform[1][3];
		temp.z=transform[2][0]*scan[i].x+transform[2][1]*scan[i].y+transform[2][2]*scan[i].z+transform[2][3];
		scan[i]=temp;
	}
}

int main(int argc, char** argv)
{
	if(argc!=10)
	{
		std::cout<<"usage: ./gicp3 file_name1 file_name2 x y z thetax thetay thetaz input_number\n";
	}
	else
	{
		std::vector<point> reading, reference;
		if(!file_read(argv[1], reading))
		{
			std::cout<<"File read failed!!!"; exit(1);
		}
		if(!file_read(argv[2], reference))
		{
			std::cout<<"File read failed!!!"; exit(1);	// write these error messages to the output file
		}
//		plot_merged(reading, reference);
		
		long double x=std::stold(argv[3]), y=std::stold(argv[4]), z=std::stold(argv[5]), thetax=std::stold(argv[6]), thetay=std::stold(argv[7]), thetaz=std::stold(argv[8]);
		std::vector<std::vector<long double>> t(4, std::vector<long double>(4, 0.0));
		std::vector<long double> eu(6); eu[0]=x; eu[1]=y; eu[2]=z; eu[3]=thetax; eu[4]=thetay; eu[5]=thetaz;
		convert(eu, t);
		process(reading, t);		
		
//		plot_merged(reading, reference);
		
		std::vector<std::vector<long double>> transformation_mat1, transformation_mat2, transformation_mat3;
		std::vector<point> ground1, ground2, rest1, rest2;
		std::vector<long double> normal1, normal2, centroid1, centroid2;
		std::vector<long double> z_axis(3, 0.0); z_axis[2]=1.0;
		std::vector<long double> origin(3, 0.0);

		
		// Find the transform to align reference plane to plane z=0 and apply
		ground_plane_extraction(reference, ground2, rest2);
		best_fit_plane1(ground2, normal2, centroid2);	// best_fit_plane
		build_transform_normal(transformation_mat1, normal2, z_axis);
		transform(reading, transformation_mat1); transform(reference, transformation_mat1);
		
		ground_plane_extraction(reference, ground2, rest2);
		best_fit_plane1(ground2, normal2, centroid2);
		std::vector<std::vector<long double>> new_transformation_mat1;
		build_transform_centroid(new_transformation_mat1, normal2, centroid2, z_axis, origin);
		transform(reading, new_transformation_mat1); transform(reference, new_transformation_mat1);
		transformation_mat1=mat_multi(new_transformation_mat1, transformation_mat1);
		
		// Find the transform to align reading plane to plane z=0 and apply
		ground_plane_extraction(reading, ground1, rest1);
		best_fit_plane1(ground1, normal1, centroid1);	// best_fit_plane
		build_transform_normal(transformation_mat2, normal1, z_axis);
		transform(reading, transformation_mat2);
		
		ground_plane_extraction(reading, ground1, rest1);
		best_fit_plane1(ground1, normal1, centroid1);
		std::vector<std::vector<long double>> new_transformation_mat2;
		build_transform_centroid(new_transformation_mat2, normal1, centroid1, z_axis, origin);
		transform(reading, new_transformation_mat2);
		transformation_mat2=mat_multi(new_transformation_mat2, transformation_mat2);
		
//		plot_merged(reading, reference);
		std::string map("all");	// change it to all
		if(!multires_registration(reading, reference, transformation_mat3, min_cell_size, hist_size, map))
		{
			std::cout<<"Incorrect 3rd argument passed"<<std::endl;
			return 1;
		}
		
		std::vector<std::vector<long double>> transformation_mat, temp;
		temp=mat_inv(transformation_mat1);
		temp=mat_multi(temp, transformation_mat3);
		temp=mat_multi(temp, transformation_mat2);
		transformation_mat=mat_multi(temp, transformation_mat1);
		transformation_mat=mat_multi(transformation_mat, t);
		
		std::string output_name(map); output_name.push_back('_');

		std::string f1(argv[1]), f2(argv[2]);

		std::string s1, s2;
		s1.push_back(f1[f1.size()-8]); s1.push_back(f1[f1.size()-7]); s1.push_back(f1[f1.size()-6]); s1.push_back(f1[f1.size()-5]);
		output_name.append(s1);
		output_name.push_back('_');
		s2.push_back(f2[f2.size()-8]); s2.push_back(f2[f2.size()-7]); s2.push_back(f2[f2.size()-6]); s2.push_back(f2[f2.size()-5]);
		output_name.append(s2);
		
		std::string input_number(argv[9]);
		output_name.push_back('_'); output_name.append(input_number);
		output_name.append("_result.txt");
		
//		file_write(output_name, transformation_mat);
		
		std::vector<long double> ans = euler_rep(transformation_mat);
		/*
		for(int i=0; i<6; ++i)
		{
			std::cout<<ans[i]<<" ";
		}
		std::cout<<std::endl;
		*/
		file_write(output_name, ans);
		std::cout<<map<<" "<<s1<<" "<<s2<<" "<<input_number<<" "<<x<<" "<<y<<" "<<z<<" "<<thetax<<" "<<thetay<<" "<<thetaz<<" "<<ans[0]<<" "<<ans[1]<<" "<<ans[2]<<" "<<ans[3]<<" "<<ans[4]<<" "<<ans[5]<<std::endl;
//		plot_merged(reading, reference);
	}
	return 0;
}
