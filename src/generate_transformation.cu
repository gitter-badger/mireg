#include "hip/hip_runtime.h"
#include <mireg.h>

bool transform_read(std::string filename, std::vector<std::vector<long double>>& transform_mat)
{
	std::ifstream file(filename);
	if(file.is_open())
	{
		for(int i=0; i<4; ++i) for(int j=0; j<4; ++j) file>>transform_mat[i][j];
		file.close();
		return true;
	}
	else
	{
		return false;
	}
}

bool transform_write(std::string filename, std::vector<std::vector<long double>>& transformation_mat)
{
	std::ofstream file(filename);
	if(file.is_open())
	{
		for(int i=0; i<4; ++i)
		{
			for(int j=0; j<4; ++j)
			{
				file<<transformation_mat[i][j]<<" ";
			}
			file<<std::endl;
		}
		file.close();
		return true;
	}
	else
	{
		return false;
	}
}

int main( int argc, char** argv)
{
	if(argc != 7)
	{
		std::cout<<"usage: ./generate_transformation x y z theta_x theta_y theta_z\n";
	}
	else
	{
		std::vector<std::vector<long double>> transform(4, std::vector<long double>(4, 0.0));
		long double x, y, z, thetax, thetay, thetaz;
		x=std::stold(argv[1]); y=std::stold(argv[2]); z=std::stold(argv[3]);
		thetax = std::stold(argv[4]); thetay = std::stold(argv[5]); thetaz = std::stold(argv[6]);
		thetax*=(pi/180.0); thetay*=(pi/180.0);	thetaz*=(pi/180.0);

		// TODO: Write the formula for conversion of euler angles to rotation matrix
		transform[0][0]=cos(thetay)*cos(thetaz);
		transform[0][1]=sin(thetax)*sin(thetay)*cos(thetaz)-cos(thetax)*sin(thetaz);
		transform[0][2]=cos(thetax)*sin(thetay)*cos(thetaz)+sin(thetax)*sin(thetaz);
		
		transform[1][0]=cos(thetay)*sin(thetaz);
		transform[1][1]=sin(thetax)*sin(thetay)*sin(thetaz)+cos(thetax)*cos(thetaz);
		transform[1][2]=cos(thetax)*sin(thetay)*sin(thetaz)-sin(thetax)*cos(thetaz);
		
		transform[2][0]=-sin(thetay);
		transform[2][1]=sin(thetax)*cos(thetay);
		transform[2][2]=cos(thetax)*cos(thetay);
		
		transform[0][3]=x;
		transform[1][3]=y;
		transform[2][3]=z;
		transform[3][3]=1.0;
		
		std::string filename("t.txt");
		transform_write(filename, transform);
	}
	return 0;
}
